
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>

__global__ void compute(int n,float* d_x, float* d_y){
    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    d_y[ti] = d_x[ti];
}

void test_cuda() {
    int n = 1000;
    int size = n*sizeof(float);

    std::vector<float> x(n);
    std::vector<float> y(n, 0);

    for(int i=0;i<n;i++){
        x[i] = i;
    }

    float* d_x;
    float* d_y;

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), size, hipMemcpyHostToDevice);

    const int block_size = 100;
    int num_blocks = (n+block_size - 1)/block_size;
    compute<<<num_blocks,block_size>>>(n, d_x, d_y);

    hipMemcpy(y.data(), d_y, size, hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++){
        std::cout << "y["<< i <<"]  : " << y[i] << std::endl;
    }

    hipFree(d_x);
    hipFree(d_y);
}

int main(int argc, char* argv[]) {
    test_cuda();
    return 0;
}